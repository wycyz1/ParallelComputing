// 系统头文件
#include <stdlib.h>
#include <stdio.h>

// cuda头文件
#include <hip/hip_runtime.h>


#define N 10 
#define GRID_SIZE 32
#define BLOCK_SIZE 16

__global__ void matrixMultiplication(float *a, float *b, float *c, int width) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float pvalue = 0;
    for (int k = 0; k < width; ++k) {
        float melement = a[ty * width + k];
        float nelement = b[k * width + tx];
        pvalue += melement * nelement;
    }
    c[ty * width + tx] = pvalue;
}


// 初始化向量为随机数值
void randomInit(float* data, unsigned int size) {
    srand(1);
    for (unsigned int i = 0; i < size; i++) {
        data[i] = rand() / (float) 100000000;
    }
}


// 主机端主函数
int main(void) {
    float *aH, *bH, *cH, *aD, *bD, *cD;
    int mem_size = N * N * sizeof(float);

    // 在主机内存申请 A，B，C 向量的空间
    aH = (float*) malloc(mem_size);
    bH = (float*) malloc(mem_size);
    cH = (float*) malloc(mem_size);

    // 在 GPU 设备申请 A，B，C 向量的空间
    hipMalloc((void**) &aD, mem_size);
    hipMalloc((void**) &bD, mem_size);
    hipMalloc((void**) &cD, mem_size);

    // 初始化主机内存的 A，B 向量
    randomInit(aH, N * N);
    randomInit(bH, N * N);

    // 拷贝主机内存的 A，B 的内容到 GPU 设备的 A，B
    hipMemcpy(aD, aH, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(bD, bH, mem_size, hipMemcpyHostToDevice);

    // GPU 内核函数的维度参数
    dim3 dimGrid(GRID_SIZE, GRID_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    // 执行 GPU 内核函数
    matrixMultiplication <<< dimGrid, dimBlock >>> (aD, bD, cD, N);

    // 从 GPU 设备复制结果向量 C 到主机内存的 C
    hipMemcpy(cH, cD, mem_size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%.2f\t", cH[i * N + j]);
        }
        printf("\n");
    }

    free(aH);
    free(bH);
    free(cH);
    hipFree(aD);
    hipFree(bD);
    hipFree(cD);
}
